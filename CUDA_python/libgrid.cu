#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>
#include <cstdio>

#include "libgrid.h"

__global__ void gridding_cuda_kernel(hipDoubleComplex *grid, double *uvw_data, hipDoubleComplex * visibility_data, double * frequency_data, int timesteps_start, int timesteps_end) {
    int timestep = timesteps_start + blockIdx.x;
    int baseline = blockIdx.y;
    int freq = threadIdx.x;

    hipDoubleComplex vis = visibility_data[(timestep * BASELINES * FREQUENCS) + (baseline * FREQUENCS) + freq];
    double frequency = frequency_data[freq];

    int iu = (int)round(THETA_OVER_C * uvw_data[(timestep * BASELINES * 3) + (baseline * 3) + 0] * frequency);
    int iv = (int)round(THETA_OVER_C * uvw_data[(timestep * BASELINES * 3) + (baseline * 3) + 1] * frequency);
    int iu_idx = iu + IMAGE_SIZE / 2;
    int iv_idx = iv + IMAGE_SIZE / 2;

    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].x), hipCreal(vis));
    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].y), hipCimag(vis));
}

extern "C" void gridding_cuda_mpi(std::complex<double> * grid, double * uvw_data, std::complex<double> * visibility_data, double * frequency_data) {
    int rank, size;
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int timesteps_per_rank = TIMESTEPS / size;
    int timesteps_start = rank * timesteps_per_rank;
    int timesteps_end = (rank == size - 1) ? TIMESTEPS : timesteps_start + timesteps_per_rank;

    hipDoubleComplex *d_grid;
    double *d_uvw_data;
    hipDoubleComplex *d_visibility_data;
    double *d_frequency_data;

    // Allocate memory on the GPU
    hipMalloc(&d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMalloc(&d_uvw_data, TIMESTEPS * BASELINES * 3 * sizeof(double));
    hipMalloc(&d_visibility_data, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex));
    hipMalloc(&d_frequency_data, FREQUENCS * sizeof(double));

    // Initialize grid on the GPU
    hipMemset(d_grid, 0, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));

    // Copy data to the GPU
    hipMemcpy(d_uvw_data, uvw_data, TIMESTEPS * BASELINES * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_visibility_data, visibility_data, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_frequency_data, frequency_data, FREQUENCS * sizeof(double), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    dim3 gridDim(timesteps_per_rank, BASELINES);
    dim3 blockDim(FREQUENCS);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    gridding_cuda_kernel<<<gridDim, blockDim>>>(d_grid, d_uvw_data, d_visibility_data, d_frequency_data, timesteps_start, timesteps_end);

    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //printf("Rank %d: CUDA kernel execution time: %f ms\n", rank, milliseconds);

    // Copy results back to the CPU
    hipDoubleComplex *h_grid = (hipDoubleComplex*) malloc(IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMemcpy(h_grid, d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // Reduce the results across all ranks
    hipDoubleComplex *global_grid = (rank == 0) ? (hipDoubleComplex*) malloc(IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex)) : nullptr;
    MPI_Reduce(h_grid, global_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), MPI_DOUBLE_COMPLEX, MPI_SUM, 0, MPI_COMM_WORLD);

    // Combine the final grid on the root rank
    if (rank == 0) {
        #pragma omp parallel for
        for (int idx = 0; idx < IMAGE_SIZE * IMAGE_SIZE; ++idx) {
            grid[idx] = std::complex<double>(hipCreal(global_grid[idx]), hipCimag(global_grid[idx]));
        }

        // Free global grid memory
        free(global_grid);
    }

    // Free GPU memory
    hipFree(d_grid);
    hipFree(d_uvw_data);
    hipFree(d_visibility_data);
    hipFree(d_frequency_data);

    // Free CPU memory
    free(h_grid);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    MPI_Finalize();
}

