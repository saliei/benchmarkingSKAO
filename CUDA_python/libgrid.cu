#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cmath>
#include <cstdio>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include "libgrid.h"

__global__ void gridding_kernel(hipDoubleComplex *grid, double *uvwt, hipDoubleComplex *vist, double *freq) {
    int timestep = blockIdx.x;
    int baseline = blockIdx.y;
    int fq = threadIdx.x;

    hipDoubleComplex vis = vist[(timestep * BASELINES * FREQUENCS) + (baseline * FREQUENCS) + fq];
    double f = freq[fq];

    int iu = (int)round(THETA_OVER_C * uvwt[(timestep * BASELINES * 3) + (baseline * 3) + 0] * f);
    int iv = (int)round(THETA_OVER_C * uvwt[(timestep * BASELINES * 3) + (baseline * 3) + 1] * f);
    int iu_idx = iu + IMAGE_SIZE_HALF;
    int iv_idx = iv + IMAGE_SIZE_HALF;

    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].x), hipCreal(vis));
    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].y), hipCimag(vis));
}

__global__ void gridding_kernel_mpi(hipDoubleComplex *grid, double *uvwt, hipDoubleComplex *vist, double *freq, int timesteps_start, int timesteps_end) {
    int timestep = timesteps_start + blockIdx.x;
    int baseline = blockIdx.y;
    int fq = threadIdx.x;

    hipDoubleComplex vis = vist[(timestep * BASELINES * FREQUENCS) + (baseline * FREQUENCS) + fq];
    double f = freq[fq];

    int iu = (int)round(THETA_OVER_C * uvwt[(timestep * BASELINES * 3) + (baseline * 3) + 0] * f);
    int iv = (int)round(THETA_OVER_C * uvwt[(timestep * BASELINES * 3) + (baseline * 3) + 1] * f);
    int iu_idx = iu + IMAGE_SIZE_HALF;
    int iv_idx = iv + IMAGE_SIZE_HALF;

    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].x), hipCreal(vis));
    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].y), hipCimag(vis));
}

void gridding_cuda(std::complex<double> *grid, double *uvwt, std::complex<double> *vist, double *freq) {
    hipDoubleComplex *d_grid;
    double *d_uvwt;
    hipDoubleComplex *d_vist;
    double *d_freq;

    hipMalloc(&d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMalloc(&d_uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double));
    hipMalloc(&d_vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex));
    hipMalloc(&d_freq, FREQUENCS * sizeof(double));

    // no need to copy the grid since it's zero initialized on the host
    hipMemset(d_grid, 0, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMemcpy(d_uvwt, uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vist, vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_freq, freq, FREQUENCS * sizeof(double), hipMemcpyHostToDevice);

    dim3 gridDim(TIMESTEPS, BASELINES);
    dim3 blockDim(FREQUENCS);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    gridding_kernel<<<gridDim, blockDim>>>(d_grid, d_uvwt d_vist, d_freq);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //printf("kernel execution time: %f ms\n", milliseconds);

    hipMemcpy(grid, d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    hipFree(d_grid);
    hipFree(d_uvwt);
    hipFree(d_vist);
    hipFree(d_freq);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// we assume a CUDA aware MPI implementation
void gridding_cuda_mpi(std::complex<double> *grid, double *uvwt, std::complex<double> *vist, double *freq) {
    int rank, size;
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int timesteps_per_rank = TIMESTEPS / size;
    int timesteps_start = rank * timesteps_per_rank;
    int timesteps_end = (rank == size - 1) ? TIMESTEPS : timesteps_start + timesteps_per_rank;

    hipDoubleComplex *d_grid;
    double *d_uvwt;
    hipDoubleComplex *d_vist;
    double *d_freq;

    hipMemset(d_grid, 0, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMalloc(&d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMalloc(&d_uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double));
    hipMalloc(&d_vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex));
    hipMalloc(&d_freq, FREQUENCS * sizeof(double));

    hipMemcpy(d_uvwt, uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vist, vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_freq, freq, FREQUENCS * sizeof(double), hipMemcpyHostToDevice);

    dim3 gridDim(timesteps_per_rank, BASELINES);
    dim3 blockDim(FREQUENCS);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    gridding_kernel_mpi<<<gridDim, blockDim>>>(d_grid, d_uvwt, d_vist, d_freq, timesteps_start, timesteps_end);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //printf("rank: %d, kernel execution time: %f ms\n", rank, milliseconds);

    //hipDoubleComplex *h_grid = (hipDoubleComplex*) malloc(IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    //hipMemcpy(h_grid, d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // reduce the results across all ranks
    hipDoubleComplex *d_grid_global = nullptr;
    if (rank == 0) {
        hipMalloc(&d_grid_global, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    }
    //hipDoubleComplex *grid_global = (rank == 0) ? (hipDoubleComplex*) malloc(IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex)) : nullptr;

    MPI_Reduce(d_grid, d_grid_global, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), MPI_DOUBLE_COMPLEX, MPI_SUM, 0, MPI_COMM_WORLD);

    if (rank == 0) {
        hipMemcpy(grid, d_grid_global, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)
    }

    hipFree(d_grid_global);
    hipFree(d_grid);
    hipFree(d_uvwt);
    hipFree(d_vist);
    hipFree(d_freq);

    //free(h_grid);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    MPI_Finalize();
}

