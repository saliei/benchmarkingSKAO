#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <complex>
#include <cmath>
#include <cstdio>

#include "libgrid.h"


__global__ void gridding_cuda_kernel(double *grid_real, double *grid_imag, double *uvw_data, double *visibility_real, double *visibility_imag, double *frequency_data) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k;

    for (k = 0; k < FREQUENCS; k++) {
        double vis_real = visibility_real[(i * BASELINES * FREQUENCS) + (j * FREQUENCS) + k];
        double vis_imag = visibility_imag[(i * BASELINES * FREQUENCS) + (j * FREQUENCS) + k];
        double freq = frequency_data[k];

        int iu = (int)round(THETA_OVER_C * uvw_data[(i * BASELINES * 3) + (j * 3) + 0] * freq);
        int iv = (int)round(THETA_OVER_C * uvw_data[(i * BASELINES * 3) + (j * 3) + 1] * freq);
        int iu_idx = iu + IMAGE_SIZE / 2;
        int iv_idx = iv + IMAGE_SIZE / 2;

        atomicAdd(&(grid_real[iu_idx * IMAGE_SIZE + iv_idx]), vis_real);
        atomicAdd(&(grid_imag[iu_idx * IMAGE_SIZE + iv_idx]), vis_imag);
    }
}


#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <complex>
#include "gridding.h"

#define IMAGE_SIZE 2048
#define IMAGE_SIZE_HALF 1024
#define THETA 0.0125
#define C 299792458
#define THETA_OVER_C 4.16955e-11

#define TIMESTEPS 512
#define BASELINES 351
#define FREQUENCS 256


__global__ void gridding_cuda_kernel(double *grid_real, double *grid_imag, double *uvw_data, double *visibility_real, double *visibility_imag, double *frequency_data) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k;

    for (k = 0; k < FREQUENCS; k++) {
        double vis_real = visibility_real[(i * BASELINES * FREQUENCS) + (j * FREQUENCS) + k];
        double vis_imag = visibility_imag[(i * BASELINES * FREQUENCS) + (j * FREQUENCS) + k];
        double freq = frequency_data[k];

        int iu = (int)round(THETA_OVER_C * uvw_data[(i * BASELINES * 3) + (j * 3) + 0] * freq);
        int iv = (int)round(THETA_OVER_C * uvw_data[(i * BASELINES * 3) + (j * 3) + 1] * freq);
        int iu_idx = iu + IMAGE_SIZE / 2;
        int iv_idx = iv + IMAGE_SIZE / 2;

        atomicAdd(&(grid_real[iu_idx * IMAGE_SIZE + iv_idx]), vis_real);
        atomicAdd(&(grid_imag[iu_idx * IMAGE_SIZE + iv_idx]), vis_imag);
    }
}

extern "C" void gridding_cuda(std::complex<double> *grid, double *uvw_data, std::complex<double> *visibility_data, double *frequency_data) {
    double *d_grid_real, *d_grid_imag;
    double *d_uvw_data;
    double *d_visibility_real, *d_visibility_imag;
    double *d_frequency_data;

    hipMalloc(&d_grid_real, IMAGE_SIZE * IMAGE_SIZE * sizeof(double));
    hipMalloc(&d_grid_imag, IMAGE_SIZE * IMAGE_SIZE * sizeof(double));
    hipMalloc(&d_uvw_data, TIMESTEPS * BASELINES * 3 * sizeof(double));
    hipMalloc(&d_visibility_real, TIMESTEPS * BASELINES * FREQUENCS * sizeof(double));
    hipMalloc(&d_visibility_imag, TIMESTEPS * BASELINES * FREQUENCS * sizeof(double));
    hipMalloc(&d_frequency_data, FREQUENCS * sizeof(double));

    double *grid_real = (double*) malloc(IMAGE_SIZE * IMAGE_SIZE * sizeof(double));
    double *grid_imag = (double*) malloc(IMAGE_SIZE * IMAGE_SIZE * sizeof(double));
    for (int idx = 0; idx < IMAGE_SIZE * IMAGE_SIZE; ++idx) {
        grid_real[idx] = std::real(grid[idx]);
        grid_imag[idx] = std::imag(grid[idx]);
    }

    double *visibility_real = (double*) malloc(TIMESTEPS * BASELINES * FREQUENCS * sizeof(double));
    double *visibility_imag = (double*) malloc(TIMESTEPS * BASELINES * FREQUENCS * sizeof(double));
    for (int idx = 0; idx < TIMESTEPS * BASELINES * FREQUENCS; ++idx) {
        visibility_real[idx] = std::real(visibility_data[idx]);
        visibility_imag[idx] = std::imag(visibility_data[idx]);
    }

    hipMemcpy(d_grid_real, grid_real, IMAGE_SIZE * IMAGE_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_imag, grid_imag, IMAGE_SIZE * IMAGE_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_uvw_data, uvw_data, TIMESTEPS * BASELINES * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_visibility_real, visibility_real, TIMESTEPS * BASELINES * FREQUENCS * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_visibility_imag, visibility_imag, TIMESTEPS * BASELINES * FREQUENCS * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_frequency_data, frequency_data, FREQUENCS * sizeof(double), hipMemcpyHostToDevice);

    gridding_cuda_kernel<<<TIMESTEPS, BASELINES>>>(d_grid_real, d_grid_imag, d_uvw_data, d_visibility_real, d_visibility_imag, d_frequency_data);

    hipMemcpy(grid_real, d_grid_real, IMAGE_SIZE * IMAGE_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(grid_imag, d_grid_imag, IMAGE_SIZE * IMAGE_SIZE * sizeof(double), hipMemcpyDeviceToHost);

    for (int idx = 0; idx < IMAGE_SIZE * IMAGE_SIZE; ++idx) {
        grid[idx] = std::complex<double>(grid_real[idx], grid_imag[idx]);
    }

    hipFree(d_grid_real);
    hipFree(d_grid_imag);
    hipFree(d_uvw_data);
    hipFree(d_visibility_real);
    hipFree(d_visibility_imag);
    hipFree(d_frequency_data);

    free(grid_real);
    free(grid_imag);
    free(visibility_real);
    free(visibility_imag);
}

