#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>
#include <cstdio>

#include "libgrid.h"


__global__ void gridding_cuda_kernel(hipDoubleComplex *grid, double *uvwt, hipDoubleComplex *vist, double *freq) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k;

    for (k = 0; k < FREQUENCS; k++) {
        hipDoubleComplex vis = vist[(i * BASELINES * FREQUENCS) + (j * FREQUENCS) + k];
        double f = freq[k];

        int iu = (int)round(THETA_OVER_C * uvwt[(i * BASELINES * 3) + (j * 3) + 0] * f);
        int iv = (int)round(THETA_OVER_C * uvwt[(i * BASELINES * 3) + (j * 3) + 1] * f);
        int iu_idx = iu + IMAGE_SIZE_HALF;
        int iv_idx = iv + IMAGE_SIZE_HALF;

        atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].x), hipCreal(vis));
        atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].y), hipCimag(vis));

    }
}

extern "C" void gridding_cuda(hipDoubleComplex *grid, double *uvwt, hipDoubleComplex *vist, double *freq) {
    hipDoubleComplex *d_grid;
    double *d_uvwt;
    hipDoubleComplex *d_vist;
    double *d_freq;

    hipMalloc(&d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMalloc(&d_uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double));
    hipMalloc(&d_vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex));
    hipMalloc(&d_freq, FREQUENCS * sizeof(double));

    hipMemcpy(d_grid, grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_uvwt, uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vist, vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_freq, freq, FREQUENCS * sizeof(double), hipMemcpyHostToDevice);

    gridding_cuda_kernel<<<TIMESTEPS, BASELINES>>>(d_grid, d_uvwt, d_vist, d_freq);

    hipMemcpy(grid, d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    hipFree(d_grid);
    hipFree(d_uvwt);
    hipFree(d_vist);
    hipFree(d_freq);
}

